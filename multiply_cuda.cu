
#include <hip/hip_runtime.h>
#include <iostream>
#include <stdio.h>
#include <time.h>
#include <math.h>

#define N 2000

using namespace std;

void fill_matrix(int *m,char c){
	cout<<"Llenamos matriz "<<endl;
	for(int i=0;i<N;i++){
		for(int j=0;j<N;j++){
			switch(c){
				case 's':
					m[i*N+j] = sin(i);break;
				case 'c':
					m[i*N+j] = cos(i);break;
				case 'z':
					m[i*N+j] = 0;break;
				default: return;
			}
		}
	}
	return;
}
//Multiplicacion en CPU
int multiply_seq(int *m1,int *m2,int *m3){
	cout<<"Multiplicamos con el algoritmo secuencial: \n"<<endl;
	for(int i=0;i<N;i++){
		for(int j=0;j<N;j++){
			for(int k=0;k<N;k++){
				m3[i*N+j]+=m1[j*N+k] * m2[k*N+i];
			}
		}
	}
	return 0;
}

//Multiplicacion en GPU
__global__ void multiply_par(int *a, int *b, int *c) {
	int k=0,suma=0;
	int i= blockIdx.x * blockDim.x + threadIdx.x;
	int j= blockIdx.y * blockDim.y + threadIdx.y;
	if(i < N && j < N){
		for(k=0;k<N;k++){
			suma+=a[j*N+k] * b[k*N+i];
		}
		c[j*N + i]= suma;
	}
}

//Imprimir matrices
void print_matrix(int *m){
	for(int i=0;i<N;i++){
		for(int j=0;j<N;j++){
			cout<<"["<< m[i*N+j] <<"]";
		}
		cout<<endl;
	}
	cout<<endl;
	return;
}

int main(){
	//Declaracion de variables
	int *h_A = (int *)malloc(N*N*sizeof(int *));;
	int *h_B = (int *)malloc(N*N*sizeof(int *));;
	int *h_C = (int *)malloc(N*N*sizeof(int *));;

	clock_t t_i,t_f;
	float tiempo;

	fill_matrix(h_A,'s');
	fill_matrix(h_B,'c');
	fill_matrix(h_C,'z');
	//print_matrix(h_A);
	//print_matrix(h_B);

	t_i=clock();
	multiply_seq(h_A,h_B,h_C);
	t_f=clock();

	tiempo= ((double)t_f - t_i) / CLOCKS_PER_SEC;
	cout<<"El tiempo de trabajo con el algoritmo secuencial es de ";
	printf("%f\n\n",tiempo);

	cout<<"Multiplicamos con el algoritmo paralelo: \n"<<endl;

	//Punteros de device
	int *d_A,*d_B,*d_C;

	int size = N * N * sizeof(int);

	hipMalloc((void **) &d_A, size);//Reservar memoria en GPU
 	hipMalloc((void **) &d_B, size);
 	hipMalloc((void **) &d_C, size);

 	hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);//Pasar datos de CPU a GPU
 	hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);

	//Invocamos el kernel
	dim3 dimBlock(N*N,2);

	t_i=clock();
	multiply_par<<<1, dimBlock>>>(d_A,d_B,d_C);
	t_f=clock();

	tiempo= ((double)t_f - t_i) / CLOCKS_PER_SEC;
	cout<<"El tiempo de trabajo con el algoritmo en paralelo es de ";
	printf("%f\n\n",tiempo);

	hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);//Pasar datos de GPU a CPU

	//print_matrix(h_C);

  hipFree(d_A);//Liberar memoria en GPU
 	hipFree(d_B);
 	hipFree(d_C);

	free(h_A);
	free(h_B);
	free(h_C);

	return 0;
}
