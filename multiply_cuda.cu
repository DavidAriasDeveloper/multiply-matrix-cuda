
#include <hip/hip_runtime.h>
#include <iostream>
#include <stdio.h>
#include <time.h>

#define N 5

using namespace std;

void fill_matrix(int m[N][N],char c){
	cout<<"Llenamos matriz: "<<endl;
	for(int i=0;i<N;i++){
		for(int j=0;j<N;j++){
			switch(c){
				case 's':
					m[i][j] = int(i+j);break;
				case 'z':
					m[i][j] = 0;break;
				default: return;
			}
		}
	}
	return;
}
//Multiplicacion en CPU
int multiply_seq(int m1[N][N], int m2[N][N],int m3[N][N]){
	cout<<"Multiplicamos con el algoritmo secuencial: \n"<<endl;
	for(int i=0;i<N;i++){
		for(int j=0;j<N;j++){
			for(int it_suma=0;it_suma<N;it_suma++){
				m3[i][j]+=m1[i][it_suma] * m2[it_suma][j];
			}
		}
	}
	return 0;
}

//Multiplicacion en GPU
__global__ void multiply_par(int *a, int *b, int *c) {
	int k=0,suma=0;
	int i= blockIdx.x * blockDim.x + threadIdx.x;
	int j= blockIdx.y * blockDim.y + threadIdx.y;
	if(i < N && j < N){
		for(k=0;k<N;k++){
			suma+=a[j*N+k] * b[k*N+i];
		}
		c[j*N + i]= suma;
	}
}

//Imprimir matrices
void print_matrix(int m[N][N]){
	for(int i=0;i<N;i++){
		for(int j=0;j<N;j++){
			cout<<"["<< m[i][j] <<"]";
		}
		cout<<endl;
	}
	cout<<endl;
	return;
}

int main(){
	//Declaracion de variables
	int matrixA[N][N];
	int matrixB[N][N];
	int matrixC[N][N];

	clock_t t_i,t_f;
	float tiempo;

	fill_matrix(matrixA,'s');
	fill_matrix(matrixB,'s');
	fill_matrix(matrixC,'z');
	print_matrix(matrixA);
	print_matrix(matrixB);

	t_i=clock();
	multiply_seq(matrixA,matrixB,matrixC);
	t_f=clock();

	tiempo= ((double)t_f - t_i) / CLOCKS_PER_SEC;
	cout<<"El tiempo de trabajo con el algoritmo secuencial es de ";
	printf("%f\n\n",tiempo);

	cout<<"Multiplicamos con el algoritmo paralelo: \n"<<endl;

	//Punteros de device
	int *d_A,*d_B,*d_C;

	int size = N * N * sizeof(int);

	hipMalloc((void **) &d_A, size);//Reservar memoria en GPU
 	hipMalloc((void **) &d_B, size);
 	hipMalloc((void **) &d_C, size);

 	hipMemcpy(d_A, matrixA, size, hipMemcpyHostToDevice);//Pasar datos de CPU a GPU
 	hipMemcpy(d_B, matrixB, size, hipMemcpyHostToDevice);

	//Invocamos el kernel
	dim3 dimBlock(N*N,2);

	t_i=clock();
	multiply_par<<<1, dimBlock>>>(d_A,d_B,d_C);
	t_f=clock();

	tiempo= ((double)t_f - t_i) / CLOCKS_PER_SEC;
	cout<<"El tiempo de trabajo con el algoritmo en paralelo es de ";
	printf("%f\n\n",tiempo);

	hipMemcpy(matrixC, d_C, size, hipMemcpyDeviceToHost);//Pasar datos de GPU a CPU

  hipFree(d_A);//Liberar memoria en GPU
 	hipFree(d_B);
 	hipFree(d_C);

	print_matrix(matrixC);
	return 0;
}
